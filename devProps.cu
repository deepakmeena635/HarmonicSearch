#include<stdio.h>

int main()
{
int devcount;
hipGetDeviceCount(&devcount);
printf("Device count:%d\n",devcount);
for (int i = 0; i < devcount; ++i){
// Get device properties
printf("\nCUDA Device #%d\n", i);
hipDeviceProp_t devProp;
hipGetDeviceProperties(&devProp, i);
printf("Name:%s\n", devProp.name);
printf("Compute capability: %d.%d\n",devProp.major ,devProp.minor);
printf("Warp Size %d\n",devProp.warpSize);
printf("Total global memory:%u bytes\n",devProp.totalGlobalMem);
printf("Total shared memory per block: %u bytes\n", devProp.sharedMemPerBlock);
printf("Total registers per block : %d\n",devProp.regsPerBlock);
printf("Clock rate: %d khz\n",devProp.clockRate);
printf("Maximum threads per block:%d\n", devProp.maxThreadsPerBlock);
for (int i = 0; i < 3; ++i)
printf("Maximum dimension %d of block: %d\n", i, devProp.maxThreadsDim[i]);
for (int i = 0; i < 3; ++i)
printf("Maximum dimension %d of grid: %d\n", i, devProp.maxGridSize[i]);
printf("Number of multiprocessors:%d\n", devProp.multiProcessorCount);
}
return 0;
}
