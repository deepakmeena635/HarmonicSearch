#include<hip/hip_runtime.h>
#include<iostream>
#include<hiprand.h>
#include<malloc.h>
#include<stdlib.h>
#include"hsutils.h"


using namespace std;




float *resolve( int dim, float low, float high,
		float brange = 0.00, int nbgood =0, int population = 100, int iterations = 10000, bool debugMode=true){
	/**
	 * res :pointer to save the result
	 * dim :dementsions of search space
	 * lower, higher bounds of serach space
	 * **/
	//TODO: suggest parameters
	//TODO: Make population dividable
	float offset= 50.00;

	float * res ;
	float rpa = 0.3; //pitch adjustment rate
	float rac = 0.8; //acceptance rate

	brange = brange >0.0 ? brange: high/100.0; //local search space range
	nbgood = nbgood>0 ? nbgood: (int)high/100; //accepted solution from 0 to nbGood-1
	
	//just to generate random number
	hiprandGenerator_t gen;
	hiprandCreateGenerator(&gen,HIPRAND_RNG_PSEUDO_DEFAULT);
	hiprandSetPseudoRandomGeneratorSeed(gen,1234ULL);

	//initializing population
	size_t pitch;
	float *harmonics = gen_random(gen, population, dim, &pitch, low, high);
	float *newHarmonics = gen_random(gen, population, dim, &pitch, low, high);
	
	//initialize accepted population
	float *bests = gen_random(gen, nbgood, dim, &pitch, low, high);
	
	// intialize second copy for updation
	float *newBest= gen_random(gen, nbgood, dim, &pitch, low,high);
	float randNo=0.0; 
	
	//configureations
	int pgrid = population>1000? 1:int(population/200)+1;
	int pblock = population>1000? population: 200 ;	
	int ggrid = nbgood>1000? 1:int(nbgood/200)+1;
	int gblock = nbgood>1000? nbgood:200;
	
	float  *obj, *sobj ;
	hipMalloc(&obj, sizeof(float)*population);
	hipMalloc(&sobj, sizeof(float)*population);
	
	float *noise = gen_random(gen, population, dim, &pitch, -1,1 );
	int *rndRecs = gen_random_indexes(gen, population, nbgood);
	
	float loss=0.0;
	float *recObj= (float*)malloc(sizeof(float)*nbgood);
	float **idek = alloc2d(nbgood, dim);
	float *merged ;
	float prevLoss = 3990000.0;
	
	//optimization loop
	for(int lolly = iterations; lolly>= 18; lolly-- ){ 	//don't quote me on this
		
		hipFree(obj);
		hipFree(sobj);
		hipFree(newBest);
		
		hipMalloc(&obj, sizeof(float)*nbgood);
		hipMalloc(&sobj, sizeof(float)*nbgood);
		newBest= gen_random(gen, nbgood, dim, &pitch, low,high);
		
		objectiveFn<<<pgrid,pblock>>>(obj, bests, dim, pitch, nbgood, offset);
		
//		//////////////////////////////
		if(debugMode){
					cout<<"before Rand"<<endl;
					hipMemcpy(recObj, obj, sizeof(float)*nbgood, hipMemcpyHostToHost);
					prnt(recObj, nbgood);
					
					hipMemcpy2D( &idek[0][0], dim*sizeof(float), bests, pitch*sizeof(float), 
													dim*sizeof(float), nbgood, hipMemcpyHostToHost);
					prnt(idek, nbgood, dim);
					
			
		}
		
//		//////////////////// 
		sorted<<<ggrid, gblock>>>(newBest, bests, obj, sobj, nbgood, dim, pitch);
		
		hipFree(bests);
		bests = newBest;
		newBest = NULL;
		
		if (debugMode){

					cout<<"sorted below"<<endl;
					hipMemcpy(recObj, sobj, sizeof(float)*nbgood, hipMemcpyHostToHost);
					prnt(recObj, nbgood);

					hipMemcpy2D( &idek[0][0], dim*sizeof(float), bests, pitch*sizeof(float), 
															dim*sizeof(float), nbgood, hipMemcpyHostToHost);
					prnt(idek, nbgood, dim);
					cout<<endl;

		}
		/////////////////////////////
		hipDeviceSynchronize();


		randNo=rand()%1000/1000.0;
		
		
	    if(randNo> rac){
	    	cout<<"Method 1: "<<lolly;
	    	hipFree(noise);
	    	hipFree(rndRecs);
	    	noise =gen_random(gen, population, dim, &pitch, -1, 1 ); 
	    	rndRecs = gen_random_indexes(gen, population, nbgood);
//	    	//////////////////////////////////////////
	    	if(debugMode){
		    	cout<<endl;
		    	free(recObj);
		    	recObj= (float*)malloc(sizeof(float)*population);
		    	hipMemcpy(recObj, rndRecs, sizeof(float)*population, hipMemcpyHostToHost);
		    	prnt(recObj, nbgood);
		    
	    		
	    	}
//	    	//////////////////////////////////////////
	    	
	    	update_harmonics<<<pgrid, pblock>>>(harmonics, bests, noise, 
	    										 rndRecs,0.0, population,dim, pitch);
	    }
	    else if (randNo> rpa){
	    	cout<<"Method 2:"<<lolly;
	    	hipFree(noise);
	    	hipFree(rndRecs);
	    	noise =gen_random(gen, population, dim, &pitch, -1, 1 ); 
	    	rndRecs = gen_random_indexes(gen, population, nbgood);
//	    	///////////////////////////////////////////
	    	if(debugMode){
				cout<<endl;
				free(recObj);
				recObj= (float*)malloc(sizeof(float)*population);
				hipMemcpy(recObj, rndRecs, sizeof(float)*population, hipMemcpyHostToHost);
				prnt(recObj, nbgood);
	    	}
//			//////////////////////////////////////////
	    	hipDeviceSynchronize();
	    	
	    	update_harmonics<<<pgrid, pblock>>>(harmonics, bests, noise, 
	    										 rndRecs,brange, population,dim, pitch);
	    }
	    else{
	    	cout<<"Method 3: "<<lolly;
	    	hipFree(harmonics);
	    	harmonics = gen_random(gen, population, dim, &pitch, low, high);
	    }
	    hipDeviceSynchronize();

	    
	    //////////////////////////////////////////
	    if(debugMode){
			cout<<endl;
			free(idek);
			idek = alloc2d(population, dim);
			
		    hipMemcpy2D( &idek[0][0], dim*sizeof(float), harmonics, 
		    					pitch*sizeof(float), dim*sizeof(float), 
		    					population, hipMemcpyDeviceToHost);
		    prnt(idek, population, dim );
		    cout<<endl;

	    }
	    //////////////////////////////////////////
	    hipDeviceSynchronize();


	    hipFree(newBest);
	    if (newHarmonics!=NULL){ 
	    	
	    	hipFree(newHarmonics);
	    }
		hipFree(obj);
		hipFree(sobj);
				
		hipMalloc(&obj, sizeof(float)*population);
		hipMalloc(&sobj, sizeof(float)*population);
		newHarmonics= gen_random(gen, population, dim, &pitch, 0.0,0.0);
		newBest= gen_random(gen, nbgood, dim, &pitch, 0.0,0.0);
		
		objectiveFn<<<pgrid,pblock>>>(obj, harmonics,dim, pitch, population, offset);
	    sorted<<<pgrid, pblock>>>(newHarmonics, harmonics, obj, sobj, population, dim, pitch);
	    hipMemcpy2D( newBest, dim*sizeof(float), newHarmonics, 
	    					pitch*sizeof(float), dim*sizeof(float), 
	    					nbgood, hipMemcpyHostToHost);
	    hipFree(newHarmonics);
	    newHarmonics = NULL;
	    
	    hipFree(obj);
	    hipMalloc(&obj, sizeof(float)*nbgood);
	    objectiveFn<<<pgrid,pblock>>>(obj, bests,dim, pitch, nbgood, offset);
	    merged = accept_better(obj, sobj, bests, newBest,nbgood, dim, pitch,  gen);
//	    hipFree(bests);
	    bests = merged;
	    
	    hipMemcpy(recObj, obj, sizeof(float)*nbgood, hipMemcpyDeviceToHost);
	    loss = avg_loss(recObj, nbgood);
//	    if (loss<prevLoss){
	    	printf("\tAVERAGE LOSS: %f\n", loss);
//	    	prevLoss = loss; 
//	    }
	   
	    if(debugMode){
	    	hipMemcpy2D( &idek[0][0], dim*sizeof(float), bests, pitch*sizeof(float), 
								dim*sizeof(float), nbgood, hipMemcpyDeviceToHost);
	    	prnt(recObj, nbgood);
	    	prnt(idek, nbgood, dim);
	    }
	    
	    
		if(debugMode){			
			system(" sleep 3");
			
		}
		if(lolly%50==0){
			system("clear");
		}
	}
	cout<<"------___---"<<endl;
	res = (float*) malloc(sizeof(float)*dim);
	hipMemcpy(res, bests, sizeof(float)*dim,hipMemcpyDeviceToHost);
	cout<<"--___----"<<endl;
	return res;
}


int main(){
	int pop= 7, dim =2002;
	int low=-2000, high =2000; 
	int nbg =100; 
	float  *result;
	int dm;
	cout<<"want debug mode (1:yes), population dim, "<<endl;
	cin>>dm;
	cin>>pop;
	cin>>dim;
	
	
	result= resolve( dim=dim,low, high, 2.0,nbg, pop, 10000, dm);
	cout<<"------"<<endl;
	//float *res, int dim, float low, float high, float brange = 0.00, int nbgood =0, int population = 100, int iterations = 10000
	cout<<endl; 
	for(int i=0; i< dim; i++)
		cout<<result[i]<<" ";
	
	return 0; 
}
