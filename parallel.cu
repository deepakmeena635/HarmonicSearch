#include<hip/hip_runtime.h>
#include<iostream>
#include<hiprand.h>
#include<malloc.h>
#include<stdlib.h>
#include"hsutils.h"
#include<chrono>


using namespace std;

float *resolve( int dim, float low, float high,
		float brange = 0.00, int nbgood =0, int population = 100, int iterations = 10000, bool debugMode=false){
	/**
	 * res :pointer to save the result
	 * dim :dementsions of search space
	 * lower, higher bounds of serach space
	 * **/
	//TODO: suggest parameters
	//TODO: Make population dividable

	auto start = std::chrono::high_resolution_clock::now();

	float offset= 50.00;

	float * res ;
	float rpa = 0.3; //pitch adjustment rate
	float rac = 0.8; //acceptance rate

	brange = brange >0.0 ? brange: high/100.0; //local search space range
	nbgood = nbgood>0 ? nbgood: (int)high/100; //accepted solution from 0 to nbGood-1
	
	//just to generate random number
	hiprandGenerator_t gen;
	hiprandCreateGenerator(&gen,HIPRAND_RNG_PSEUDO_DEFAULT);
	hiprandSetPseudoRandomGeneratorSeed(gen,1234ULL);

	//initializing population
	size_t pitch;
	float *harmonics = gen_random(gen, population, dim, &pitch, low, high);
	float *newHarmonics = gen_random(gen, population, dim, &pitch, low, high);
	
	//initialize accepted population
	float *bests = gen_random(gen, nbgood, dim, &pitch, low, high);
	
	// intialize second copy for updation
	float *newBest= gen_random(gen, nbgood, dim, &pitch, low,high);
	float randNo=0.0; 
	
	//configureations
	int pgrid = population>1024? 1:int(population/512)+1;
	int pblock = population>1024? population: 512 ;	
	int ggrid = nbgood>1024? 1:int(nbgood/512)+1;
	int gblock = nbgood>1024? nbgood:512;
	
	float  *obj, *sobj ;
	hipMalloc(&obj, sizeof(float)*population);
	hipMalloc(&sobj, sizeof(float)*population);
	
	float *noise = gen_random(gen, population, dim, &pitch, -1,1 );
	int *rndRecs = gen_random_indexes(gen, population, nbgood);
	
	float loss=0.0;
	float *recObj= (float*)malloc(sizeof(float)*nbgood);
	float **idek = alloc2d(nbgood, dim);
	float *merged ;
	float prevLoss = 3990000.0;
	int choise =0; 
	//optimization loop
	for(int lolly = iterations; lolly>= 18; lolly-- ){ 	//don't quote me on this
		
		hipFree(obj);
		hipFree(sobj);
		hipFree(newBest);
		
		hipMalloc(&obj, sizeof(float)*nbgood);
		hipMalloc(&sobj, sizeof(float)*nbgood);
		newBest= gen_random(gen, nbgood, dim, &pitch, low,high);
		
		objectiveFn<<<pgrid,pblock>>>(obj, bests, dim, pitch, nbgood, offset);
		
//		//////////////////////////////
		if(debugMode ){
					cout<<"before Rand"<<endl;
					hipMemcpy(recObj, obj, sizeof(float)*nbgood, hipMemcpyHostToHost);
					prnt(recObj, nbgood);
					
					hipMemcpy2D( &idek[0][0], dim*sizeof(float), bests, pitch*sizeof(float), 
													dim*sizeof(float), nbgood, hipMemcpyHostToHost);
					prnt(idek, nbgood, dim);
					
			
		}
		
//		//////////////////// 
		sorted<<<ggrid, gblock>>>(newBest, bests, obj, sobj, nbgood, dim, pitch);
		
		hipFree(bests);
		bests = newBest;
		newBest = NULL;
		////////////////////////////////////////
		if (debugMode && false){

					cout<<"sorted below"<<endl;
					hipMemcpy(recObj, sobj, sizeof(float)*nbgood, hipMemcpyHostToHost);
					prnt(recObj, nbgood);

					hipMemcpy2D( &idek[0][0], dim*sizeof(float), bests, pitch*sizeof(float), 
															dim*sizeof(float), nbgood, hipMemcpyHostToHost);
					prnt(idek, nbgood, dim);
					cout<<endl;

		}
		/////////////////////////////
		hipDeviceSynchronize();

		randNo=rand()%1000/1000.0;
		
		
	    if(randNo> rac){
	    	choise =1;
	    	hipFree(noise);
	    	hipFree(rndRecs);
	    	noise =gen_random(gen, population, dim, &pitch, -1, 1 ); 
	    	rndRecs = gen_random_indexes(gen, population, nbgood);
//	    	//////////////////////////////////////////
	    	if(debugMode && false){
		    	cout<<endl;
		    	free(recObj);
		    	recObj= (float*)malloc(sizeof(float)*population);
		    	hipMemcpy(recObj, rndRecs, sizeof(float)*population, hipMemcpyHostToHost);
		    	prnt(recObj, nbgood);
	    	}
//	    	//////////////////////////////////////////
	    	
	    	update_harmonics<<<pgrid, pblock>>>(harmonics, bests, noise, 
	    										 rndRecs,0.0, population,dim, pitch);
	    }
	    else if (randNo> rpa){
	    	choise =2;
	    	hipFree(noise);
	    	hipFree(rndRecs);
	    	noise =gen_random(gen, population, dim, &pitch, -1, 1 ); 
	    	rndRecs = gen_random_indexes(gen, population, nbgood);
//	    	///////////////////////////////////////////
	    	if(debugMode && false){
				cout<<endl;
				free(recObj);
				recObj= (float*)malloc(sizeof(float)*population);
				hipMemcpy(recObj, rndRecs, sizeof(float)*population, hipMemcpyHostToHost);
				prnt(recObj, nbgood);
	    	}
//			//////////////////////////////////////////
	    	hipDeviceSynchronize();
	    	
	    	update_harmonics<<<pgrid, pblock>>>(harmonics, bests, noise, 
	    										 rndRecs,brange, population,dim, pitch);
	    }
	    else{
	    	choise =3;
	    	hipFree(harmonics);
	    	harmonics = gen_random(gen, population, dim, &pitch, low, high);
	    }
	    hipDeviceSynchronize();

	    
	    //////////////////////////////////////////
	    if(debugMode && false){
			cout<<endl;
			free(idek);
			idek = alloc2d(population, dim);
			
		    hipMemcpy2D( &idek[0][0], dim*sizeof(float), harmonics, 
		    					pitch*sizeof(float), dim*sizeof(float), 
		    					population, hipMemcpyDeviceToHost);
		    prnt(idek, nbgood, dim );// only first nbgood 
		    cout<<endl;

	    }
	    //////////////////////////////////////////
	    hipDeviceSynchronize();


	    hipFree(newBest);
	    if (newHarmonics!=NULL){ 
	    	
	    	hipFree(newHarmonics);
	    }
		hipFree(obj);
		hipFree(sobj);
				
		hipMalloc(&obj, sizeof(float)*population);
		hipMalloc(&sobj, sizeof(float)*population);
		newHarmonics= gen_random(gen, population, dim, &pitch, 0.0,0.0);
		newBest= gen_random(gen, nbgood, dim, &pitch, 0.0,0.0);
		
		objectiveFn<<<pgrid,pblock>>>(obj, harmonics,dim, pitch, population, offset);
	    sorted<<<pgrid, pblock>>>(newHarmonics, harmonics, obj, sobj, population, dim, pitch);
	    hipMemcpy2D( newBest, dim*sizeof(float), newHarmonics, 
	    					pitch*sizeof(float), dim*sizeof(float), 
	    					nbgood, hipMemcpyHostToHost);
	    hipFree(newHarmonics);
	    newHarmonics = NULL;
	    
	    hipFree(obj);
	    hipMalloc(&obj, sizeof(float)*nbgood);
	    objectiveFn<<<pgrid,pblock>>>(obj, bests,dim, pitch, nbgood, offset);
	    merged = accept_better(obj, sobj, bests, newBest,nbgood, dim, pitch,  gen);
//	    hipFree(bests);
	    bests = merged;
	    
	    hipMemcpy(recObj, obj, sizeof(float)*nbgood, hipMemcpyDeviceToHost);
	    loss = avg_loss(recObj, nbgood);
	    if (true ){
	    	printf("%d choise:%d   AVERAGE LOSS: %f\n", lolly, choise, loss);
	    	prevLoss = loss; 
	    }
	   
	    if(debugMode && false){
	    	hipMemcpy2D( &idek[0][0], dim*sizeof(float), bests, pitch*sizeof(float), 
								dim*sizeof(float), nbgood, hipMemcpyDeviceToHost);
	    	prnt(recObj, nbgood);
	    	prnt(idek, nbgood, dim);
	    }
	    
	    
		//if(debugMode){			
	//		system(" sleep 3");
			
		//}
	
		if (loss< nbgood*0.1 ){
				break;
		}
	}
	auto end = std::chrono::high_resolution_clock::now();
	double time_elapsed = double(std::chrono::duration_cast<std::chrono::milliseconds>(end - start).count());
	std::cout << std::endl << "Elapsed Time(s): " << time_elapsed /1000<< std::endl;
	
	res = (float*) malloc(sizeof(float)*dim);
	hipMemcpy(res, bests, sizeof(float)*dim,hipMemcpyDeviceToHost);
	
	return res;
}


int main(){
	
	int iterations = 10000;
	int dim = 128;
	int low=-2000, high =2000; 	
	int nbg = 3;
	int pop = 512; //population
	float  *result;
	int dm=0;
	
	nbg = (int) pop/10 ;
	cout<<" population: "<<pop<<" nb_g: "<<nbg<<endl;

	result = resolve(dim, low, high, 0.2, nbg, pop, iterations, dm);
	for( int i=0; i< dim; i++){
		cout<<result[i]<<" ";
	}	
	
	
	return 0; 
}
