#include "hip/hip_runtime.h"
#include<hip/hip_runtime.h>
#include<iostream>
#include<hiprand.h>
#include<malloc.h>
#include<stdlib.h>
#include"hsutils.h"

using namespace std;


__global__ void idek(float *resHarms, float* harmonics, 
					float* obj, int pop,int  dim,size_t  pitch){

	int id = threadIdx.x; 
	if(id > pop){
		return ;
	}
	obj[id]= id;
}

void test_gen_random(int pop, int dim, float low, float high){

	printf("--------Random gen and vector_scale and inrange tested together -------\n");
		
	hiprandGenerator_t gen;
	hiprandCreateGenerator(&gen, HIPRAND_RNG_PSEUDO_DEFAULT);
	hiprandSetPseudoRandomGeneratorSeed(gen,551234ULL);
	size_t pitch; 
	
	
	for (int i=0 ; i< 4 ; i++){
	
		float* arr = gen_random(gen, pop, dim, &pitch, low, high);
		float brr[pop][dim];
		hipMemcpy2D(brr, dim*sizeof(float), arr, pitch*sizeof(float), dim*sizeof(float),pop, hipMemcpyDeviceToHost);
		
		for(int i=0; i< 10; i++){
			for(int j=0 ; j< dim; j++){
				cout<<" "<<brr[i][j];
			}
			cout<<endl; 
		}
			
	}
		printf("----------------DONE////////////////////////::::::::-__-__-__-\n");
}

void test_alloc2d(){

	printf("---------------------alloc2d test-----------------\n");
	int n= 7, m=5;
	float *arr =(float *) malloc(sizeof(float)*m*n);
	for(int i=0; i<m*n;i++){
		arr[i]=i;
	}
	float **brr = alloc2d(m,n, arr);
	printf("assigninng 0, m*n to arr\n");	
	prnt(brr, m,n); 
	printf("-----------------alloc2dWorks------------------\n");
}


void  test_swap(){
	
	float  *a, *b ;
	float inpa[] = {0.9, 1.0, 1.2, 1.3, 1.4, 1.5, 0.9, 1.0, 1.2, 1.3, 1.4, 1.5};
	float inpb[] = {0.1, 0.2, 0.3, 0.4, 0.1, 0.2, 0.3, 0.4, 0.1, 0.2, 0.3, 0.4 };
	hipMalloc(&a,sizeof(float)*12 );
	hipMalloc(&b,sizeof(float)*12 );
	
	hipMemcpy(a, inpa, sizeof(float)*12, hipMemcpyHostToDevice);
	hipMemcpy(b, inpb, sizeof(float)*12, hipMemcpyHostToDevice);
	
	swap<<<3,3>>>(a,b,1,3,7);
	
	hipMemcpy(inpa, a ,sizeof(float)*12, hipMemcpyDeviceToHost);
	hipMemcpy(inpb, b, sizeof(float)*12, hipMemcpyDeviceToHost);
	
	prnt(inpa, 12 );
	prnt(inpb, 12 );
	printf("-----------------swap end------------------\n");
	
}



void test_objectiveFn(int pop, int col, int grd, int blk, hiprandGenerator_t gen){
	printf("-----------------testing objective function------------------\n");

	//float *res, float *harmonics, int dim, int pitch, int population
	/**
		float *pop, size_t pitch, int low, int high, int population
	**/


	float *res, *harmonics;
	size_t pitch; 
	
	// generate random harmioncs in range and display
	hipMallocPitch(&harmonics, &pitch, sizeof(float)*col, sizeof(float)*pop);
	hiprandGenerateUniform(gen, harmonics, pitch*pop);
	inrange<<<1,pop>>>(harmonics, pitch, -100.0, 100.0, pop, col);
	
	float** hostHarm = alloc2d(pop, col);	
	hipMemcpy2D(&hostHarm[0][0], col*sizeof(float), harmonics, 
				pitch*sizeof(float), col*sizeof(float), pop, hipMemcpyDeviceToHost);
	
	// get objective values
	hipMalloc(&res, sizeof(float)*pop);
	objectiveFn<<<1,pop>>>(res, harmonics, col, pitch, pop, 67);
	
	float *disp =(float*) malloc(sizeof(float)*pop);
	hipMemcpy(disp, res , sizeof(float)*pop, hipMemcpyDeviceToHost);

	cout<<"------out"<<endl;
	//free(disp);
	prnt(hostHarm, pop, col);
	
	prnt(disp, pop);
	printf("-----------------objective function ends------------------\n");
}


void test_sorted(){

	hiprandGenerator_t gen;
	hiprandCreateGenerator(&gen, HIPRAND_RNG_PSEUDO_DEFAULT);
	hiprandSetPseudoRandomGeneratorSeed(gen,1234ULL);
	size_t pitch; 
	
	int pop = 1001;
	int dim = 1001;
	int col = dim;
	float *obj, *sobj ;
	
	//generate 
	float *harmonics = gen_random(gen, pop, dim, &pitch, -10, 10);
	float *resHarms = gen_random(gen, pop, dim, &pitch, 0.0,0.0);
	
	hipMalloc(&obj, sizeof(float)*pop);
	hipMalloc(&sobj, sizeof(float)*pop);
	objectiveFn<<<1,pop>>>(obj, harmonics, col, pitch, pop, 69);
	
	//print harmonics
	float brr[pop][dim];
	hipMemcpy2D( brr, dim*sizeof(float), harmonics, pitch*sizeof(float), dim*sizeof(float),pop, hipMemcpyDeviceToHost);
	
	
	
	
	//idek<<<1,pop>>>(resHarms, harmonics, obj,sobj,  pop, dim, pitch);
	
	sorted<<<(int)pop/3+1,3>>>( resHarms, harmonics, obj,sobj,  pop, dim, pitch);
	hipDeviceSynchronize();
	 
	//display objective values
	float *disp =(float*) malloc(sizeof(float)*pop);
	
	CUDA_CALL(hipMemcpy(disp, obj, sizeof(float)*pop, hipMemcpyDeviceToHost));
	//prnt(disp, pop);
	CUDA_CALL(hipMemcpy(disp, sobj, sizeof(float)*pop, hipMemcpyDeviceToHost));
	prnt(disp, pop);
	
	gpuErrchk(hipMemcpy2D( brr, dim*sizeof(float), resHarms, pitch*sizeof(float), 
				dim*sizeof(float), pop, hipMemcpyDeviceToHost));
	
		
}

void test_update_harmonics(){
	
	cout<<"-------------------testing update harmonics------------"<<endl;
	hiprandGenerator_t gen;
	hiprandCreateGenerator(&gen, HIPRAND_RNG_PSEUDO_DEFAULT);
	hiprandSetPseudoRandomGeneratorSeed(gen,1234ULL);
	size_t pitch; 
	
	int pop = 10;
	int dim = 3;
	int col = dim;
	
	float brange = 2.0;
	int nbgood = 3; 
	
	
	//generate 
	float *harmonics= gen_random(gen, pop, dim, &pitch, 2, 10);
	float *best = gen_random(gen, nbgood, dim, &pitch,0,2 );
	float *noise =gen_random(gen, pop, dim, &pitch, -1,1 ); 

	float** hostHarm = alloc2d(pop, col);
	hipMemcpy2D(&hostHarm[0][0], col*sizeof(float), harmonics, 
			pitch*sizeof(float), col*sizeof(float), pop, hipMemcpyDeviceToHost);
	
	cout<<"-----------harmonics intially-----------"<<endl;
	prnt(hostHarm, pop, dim);
	
	cout<<"-----------best hamrmonics-----------"<<endl;
	float** bestHrms= alloc2d(nbgood, col);
	hipMemcpy2D(&bestHrms[0][0], col*sizeof(float), best, 
			pitch*sizeof(float), col*sizeof(float), nbgood, hipMemcpyDeviceToHost);
	
	prnt(bestHrms, nbgood, dim);	
	
	int recs[] = {0,1,2,0,1,2,0,1,2,0};
	int *rndRecs;
	hipMalloc(&rndRecs, sizeof(int)*pop );
	hipMemcpy(rndRecs, recs, sizeof(int)*pop, hipMemcpyHostToDevice);
	update_harmonics<<<1, pop >>>(harmonics, best, noise, rndRecs, brange, pop, dim, pitch);
	
	//int col = dim ;
		
	hipMemcpy2D(&hostHarm[0][0], col*sizeof(float), harmonics, 
				pitch*sizeof(float), col*sizeof(float), pop, hipMemcpyDeviceToHost);
	cout<<"-----------after update-----------"<<endl;
	prnt(hostHarm, pop, dim);
	
	cout<<"-------------------update harmonics testing done------------"<<endl;
	
}

void test_random_indexes(hiprandGenerator_t gen  ){
	//hiprandGenerator_t gen, int row, int maxIndex
	int rows = 100, maxIndex=5;
	
	int *indexes = gen_random_indexes(gen,rows,maxIndex);
	
	int* res = (int*) malloc(sizeof(int)*rows);
		
	hipMemcpy(res, indexes, sizeof(int )*rows, hipMemcpyDeviceToHost);
	for(int i=0; i<rows;i++){
		cout<<" "<<res[i];
	}
		
}

void test_accept_better( hiprandGenerator_t gen){
	///		accept_better(float*obj, float*sobj, float*bests, 
	// 		float* newBests, int nbgood, int dim, size_t pitch
	
	int pop= 6, col=3;
	int dim =col, nbgood = 4; 
	float *obj, *sobj;
	size_t pitch; 
	int population =pop;

	hipMalloc(&obj, sizeof(float)*pop); 
	hipMalloc(&sobj, sizeof(float)*pop);

	float * sample1, *sample2;

	sample1 = gen_random(gen, pop, dim, &pitch, -20, 20);
	sample2 = gen_random(gen, pop, dim, &pitch, -10, 10);

	objectiveFn<<<1,pop>>>(obj, sample1, col, pitch, pop, 67);
	objectiveFn<<<1,pop>>>(sobj, sample2, col, pitch, pop, 67);

	float *res1, *res2;
	res1 = (float*) malloc(sizeof(float)*pop);
	res2 = (float*) malloc(sizeof(float)*pop);

	hipMemcpy(res1, obj, sizeof(float)*pop, hipMemcpyDeviceToHost );
	hipMemcpy(res2, sobj, sizeof(float)*pop, hipMemcpyDeviceToHost);

	prnt(res1, pop);
	prnt(res2, pop);

	float * nsample1, *nsample2 ; 

	nsample1 = gen_random(gen, pop, dim, &pitch, -20, 15);
	nsample2 = gen_random(gen, pop, dim, &pitch, -10, 10);
	
	float *nobj, *nsobj; 
	
	hipMalloc(&nobj, sizeof(float)*pop); 
	hipMalloc(&nsobj, sizeof(float)*pop);
	cout<<"---------------------------\n";
	
	sorted<<<1,pop>>>(	nsample1, sample1, obj, nobj, pop, dim, pitch);
	sorted<<<1,pop>>>(	nsample2, sample2, sobj, nsobj, pop, dim, pitch);
	
	hipMemcpy(res1, nobj, sizeof(float)*pop, hipMemcpyDeviceToHost );
	hipMemcpy(res2, nsobj, sizeof(float)*pop, hipMemcpyDeviceToHost);

	prnt(res1, pop);
	prnt(res2, pop);
	
	
	///////////////////////////////////////////////////////////////////////////////////
	
	float  **idek = alloc2d(population, dim);
	
    hipMemcpy2D( &idek[0][0], dim*sizeof(float), nsample1, 
    					pitch*sizeof(float), dim*sizeof(float), 
    					population, hipMemcpyDeviceToHost);
    cout<<"---------------------------\n";
    prnt(idek, population, dim);
	idek = alloc2d(pop, dim);
	
    hipMemcpy2D( &idek[0][0], dim*sizeof(float), nsample2, 
    					pitch*sizeof(float), dim*sizeof(float), 
    					population, hipMemcpyDeviceToHost);
    cout<<"---------------------------\n";
    prnt(idek, population, dim);
    cout<<"------------pitch---"<<pitch<<endl;
	float *ress =  accept_better(nobj, nsobj, nsample1, nsample2, nbgood, dim, pitch, gen); //<---------------
	
	
	free(idek);
	idek = alloc2d(pop, dim);

    hipMemcpy2D( &idek[0][0], dim*sizeof(float), ress, 
  					pitch*sizeof(float), dim*sizeof(float), 
					nbgood, hipMemcpyDeviceToHost);
	cout<<"---------------------------\n";
    prnt(idek, nbgood, dim);

	///////////////////////////////////////////////////////////////////////
	hipMemcpy(res1, nobj, sizeof(float)*pop, hipMemcpyDeviceToHost);
	hipMemcpy(res2, nsobj, sizeof(float)*pop, hipMemcpyDeviceToHost);

	cout<<"---------------------------\n";
	prnt(res1, pop);
	prnt(res2, pop);	
}



int main(){

	int pop = 2048, col = 5;
	//test_alloc2d();
		
	//////////////////////GEN RANDOM ///////////////////////
	test_gen_random(pop, col, -1000.0, 1000.0);
	
	///////////////////////OBJFN////////////////////////////
	hiprandGenerator_t gen;
	hiprandCreateGenerator(&gen, HIPRAND_RNG_PSEUDO_DEFAULT);
	hiprandSetPseudoRandomGeneratorSeed(gen,1234ULL);
	//size_t pitch; 
	//test_objectiveFn(pop, col, 4, 3, gen);	
	
	//////////////////////SWAP//////////////////////////////
	//test_swap();
	//test_sorted();
	//test_update_harmonics();
	//test_random_indexes(gen);
	//test_accept_better(gen);
	return 0;
}
