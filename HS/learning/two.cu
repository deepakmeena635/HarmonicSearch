#include<hip/hip_runtime.h>
#include<iostream>
#include<hiprand.h>
#include<malloc.h>
#include<stdlib.h>
#include"hsutils.h"


using namespace std;




void resolve(float *res, int dim, float low, float high,
		float brange = 0.00, int nbgood =1, int population = 100, int iterations = 10000){
	/**
	 * res :pointer to save the result
	 * dim :dementsions of search space
	 * lower, higher bounds of serach space
	 * **/
	//TODO: suggest parameters
	//TODO: Make population dividable

	int population = 50; //number of harmonics
	int dim = sol_dims;  //search space dimensions

	float rpa = 0.3; //pitch adjustment rate
	float rac = 0.9; //acceptance rate

	brange = brange >0.0 ? brange:high/100.0; //local search space range
	nbgood = nbgood>0 ? nbgood: (int)high/10; //accepted solution from 0 to nbGood-1
	float prev_loss = 91111111.0;

	//just to generate random number
	hiprandGenerator_t gen;
	hiprandCreateGenerator(&gen,HIPRAND_RNG_PSEUDO_DEFAULT);
	hiprandSetPseudoRandomGeneratorSeed(gen,1234ULL);

	//initializing population
	size_t pitch;
	float *harmonics = gen_random(gen, population, dim, &pitch, low, high);
	
	//initialize accepted population
	float **bests = gen_random(gen, nbgood, dim, &pitch, low, high);
	
	// intialize second copy for updation
	float **newBset= gen_random(gen, nbgood, dim, &pitch, low, high);
	float rand =0.0; 
	//optimization loop
	for(int lolly = iterations; lolly>= 18; lolly-- ){ 	//don't quote me on this

		bests = sorted<<<1,1>>>(bests, dim+1);
		rand = 1.0/1.0+(float)rand()%100;
	    if(rand > rac){
	    	updated_harmonics(harmonics, nbgood, 0);
	    }
	    else if rand > rpa:
	        nhrs = updated_harmonics(harmonics, nbgood, brange)
	    else:
//	        choices.append(3)
	        nhrs = gen(harmonics, nbgood, low, high)

	    nhrs = sorted(nhrs, key= lambda x: obj(*x))
	    loss = sum([obj(*i) for i in nhrs[:nbgood] ])

	    if loss< prev_loss:
	        harmonics = nhrs
	        prev_loss = loss
//	    losses.append(loss)
	}
}









int main(){
	int m = 20, n =3;
	int low=-200, high =200; 
	float *ad, *best;
	size_t pitch=NULL;
	
	//prepare generator once n 4 all
	hiprandGenerator_t gen;
	hiprandCreateGenerator(&gen, HIPRAND_RNG_PSEUDO_DEFAULT);
	hiprandSetPseudoRandomGeneratorSeed(gen,1234ULL);
		
	/* Allocate m*n random floats on device */
	ad = gen_random(gen, m, n, &pitch, low, high);
	
	float hst[m][n];
	hipMemcpy2D(hst, n*sizeof(float), ad, pitch*sizeof(float), n*sizeof(float), m, hipMemcpyDeviceToHost);

	cout<<(sizeof(hst))/sizeof(float)<<endl;
	for(int i=0 ; i< m; i++)
	{
		for(int j=0 ; j<n; j++){
			cout<<hst[i][j]<<' ';			
		}
		cout<<endl;
	}
	cout<<"--------"<<endl;
	return 0; 
}
