#include "hip/hip_runtime.h"
#include<stdio.h>
#include<hip/hip_runtime.h>
#include<hip/hip_runtime.h>
#include<>
#include<iostream>

//#include "Utilities.cuh"
using namespace std ; 

#define BLOCKSIZE_x 16
#define BLOCKSIZE_y 16

#define Nrows 3
#define Ncols 5

int iDivUp(int a, int b){ 
	return ((a % b) != 0) ? (a / b + 1) : (a / b); 
}



/******************/
/* TEST KERNEL 2D */
/******************/
__global__ void test_kernel_2D(float *devPtr, size_t pitch)
{
	int    tidx = blockIdx.x*blockDim.x + threadIdx.x;
	int    tidy = blockIdx.y*blockDim.y + threadIdx.y;

	if ((tidx < Ncols) && (tidy < Nrows))
	{
		float *row_a = (float *)((char*)devPtr + tidy * pitch);
		row_a[tidx] = row_a[tidx] * tidx * tidy;
	}
}

/********/
/* MAIN */
/********/
int main()
{
	float hostPtr[Nrows][Ncols];
	float *devPtr;
	size_t pitch;

	for (int i = 0; i < Nrows; i++)
		for (int j = 0; j < Ncols; j++) {
			hostPtr[i][j] = 1.f;
			//printf("row %i column %i value %f \n", i, j, hostPtr[i][j]);
		}

	// --- 2D pitched allocation and host->device memcopy
	(hipMallocPitch(&devPtr, &pitch, Ncols * sizeof(float), Nrows));
	(hipMemcpy2D(devPtr, pitch, hostPtr, Ncols*sizeof(float), Ncols*sizeof(float), Nrows, hipMemcpyHostToDevice));

	dim3 gridSize(iDivUp(Ncols, BLOCKSIZE_x), iDivUp(Nrows, BLOCKSIZE_y));
	dim3 blockSize(BLOCKSIZE_y, BLOCKSIZE_x);

	test_kernel_2D << <gridSize, blockSize >> >(devPtr, pitch);
	(hipPeekAtLastError());
	(hipDeviceSynchronize());

	(hipMemcpy2D(hostPtr, Ncols * sizeof(float), devPtr, pitch, Ncols * sizeof(float), Nrows, hipMemcpyDeviceToHost));

	for (int i = 0; i < Nrows; i++){
		for (int j = 0; j < Ncols; j++){
			printf("%f ", i, j, hostPtr[i][j]);
			}
		cout<<endl;
	}
	return 0;

}
