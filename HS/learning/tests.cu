#include<hip/hip_runtime.h>
#include<iostream>
#include<hiprand.h>
#include<malloc.h>
#include<stdlib.h>
#include"hsutils.h"

using namespace std;


void test_gen_random(int pop, int dim, float low, float high){

	printf("--------Random gen and vector_scale and inrange tested together -------\n");
		
	hiprandGenerator_t gen;
	hiprandCreateGenerator(&gen, HIPRAND_RNG_PSEUDO_DEFAULT);
	hiprandSetPseudoRandomGeneratorSeed(gen,1234ULL);
	size_t pitch; 
	
	float* arr = gen_random(gen, pop, dim, &pitch, low, high);
	
	float brr[pop][dim];
	hipMemcpy2D(brr, dim*sizeof(float), arr, pitch*sizeof(float), dim*sizeof(float),pop, hipMemcpyDeviceToHost);
	
	for(int i=0; i< pop; i++){
		for(int j=0 ; j< dim; j++){
			cout<<" "<<brr[i][j];
		}
		cout<<endl; 
	}
	printf("----------------DONE////////////////////////::::::::-__-__-__-\n");
}

void test_alloc2d(){

	printf("---------------------alloc2d test-----------------\n");
	int n= 7, m=5;
	float *arr =(float *) malloc(sizeof(float)*m*n);
	for(int i=0; i<m*n;i++){
		arr[i]=i;
	}
	float **brr = alloc2d(m,n, arr);
	printf("assigninng 0, m*n to arr\n");	
	prnt(brr, m,n); 
	printf("-----------------alloc2dWorks------------------\n");
}


void  test_swap(){
	
	float  *a, *b ;
	float inpa[] = {0.9, 1.0, 1.2, 1.3, 1.4, 1.5, 0.9, 1.0, 1.2, 1.3, 1.4, 1.5};
	float inpb[] = {0.1, 0.2, 0.3, 0.4, 0.1, 0.2, 0.3, 0.4, 0.1, 0.2, 0.3, 0.4 };
	hipMalloc(&a,sizeof(float)*12 );
	hipMalloc(&b,sizeof(float)*12 );
	
	hipMemcpy(a, inpa, sizeof(float)*12, hipMemcpyHostToDevice);
	hipMemcpy(b, inpb, sizeof(float)*12, hipMemcpyHostToDevice);
	
	swap<<<3,3>>>(a,b,1,3,7);
	
	hipMemcpy(inpa, a ,sizeof(float)*12, hipMemcpyDeviceToHost);
	hipMemcpy(inpb, b, sizeof(float)*12, hipMemcpyDeviceToHost);
	
	prnt(inpa, 12 );
	prnt(inpb, 12 );
	printf("-----------------swap end------------------\n");
	
}



void test_objectiveFn(int pop, int col, int grd, int blk, hiprandGenerator_t gen){
	printf("-----------------testing objective function------------------\n");

	//float *res, float *harmonics, int dim, int pitch, int population
	/**
		float *pop, size_t pitch, int low, int high, int population
	**/


	float *res, *harmonics;
	size_t pitch; 
	
	// generate random harmioncs in range and display
	hipMallocPitch(&harmonics, &pitch, sizeof(float)*col, sizeof(float)*pop);
	hiprandGenerateUniform(gen, harmonics, pitch*pop);
	inrange<<<1,pop>>>(harmonics, pitch, -100.0, 100.0, pop, col);
	
	float** hostHarm = alloc2d(pop, col);	
	hipMemcpy2D(&hostHarm[0][0], col*sizeof(float), harmonics, 
				pitch*sizeof(float), col*sizeof(float), pop, hipMemcpyDeviceToHost);
	
	// get objective values
	hipMalloc(&res, sizeof(float)*pop);
	objectiveFn<<<1,pop>>>(res, harmonics, col, pitch, pop);
	
	float *disp =(float*) malloc(sizeof(float)*pop);
	hipMemcpy(disp, res , sizeof(float)*pop, hipMemcpyDeviceToHost);

	cout<<"------out"<<endl;
	//free(disp);
	prnt(hostHarm, pop, col);
	
	prnt(disp, pop);
	printf("-----------------objective function ends------------------\n");
}


void test_sorted(){

	hiprandGenerator_t gen;
	hiprandCreateGenerator(&gen, HIPRAND_RNG_PSEUDO_DEFAULT);
	hiprandSetPseudoRandomGeneratorSeed(gen,1234ULL);
	size_t pitch; 
	
	int pop = 10;
	int dim = 5;
	int col = dim;
	float *obj ;
	
	//generate 
	float *harmonics = gen_random(gen, pop, dim, &pitch, -10, 10);
	float *resHarms = gen_random(gen, pop, dim, &pitch, 0.0,0.0);
	
	hipMalloc(&obj, sizeof(float)*pop);
	objectiveFn<<<1,pop>>>(obj, harmonics, col, pitch, pop);
	
	//print harmonics
	
	float brr[pop][dim];
	hipMemcpy2D( brr, dim*sizeof(float), harmonics, pitch*sizeof(float), dim*sizeof(float),pop, hipMemcpyDeviceToHost);
	for(int i=0; i< pop; i++){
		for(int j=0 ; j< dim; j++){
			cout<<" "<<brr[i][j];
		}
		cout<<endl; 
	}
		
	sorted<<<1, pop>>>( resHarms , harmonics, obj, pop, dim, pitch);

	//display objective function
	float *disp =(float*) malloc(sizeof(float)*pop);
	hipMemcpy(disp, obj, sizeof(float)*pop, hipMemcpyDeviceToHost);
	prnt(disp, pop);
					
	//float brr[pop][dim];
	hipMemcpy2D( brr, dim*sizeof(float), resHarms, pitch*sizeof(float), dim*sizeof(float),pop, hipMemcpyDeviceToHost);
	for(int i=0; i< pop; i++){
		for(int j=0 ; j< dim; j++){
			cout<<" "<<brr[i][j];
		}
		cout<<endl; 
	}
	
	
	
}

void test_harmonic_update(){}

void test_update_harmonics(){}


int main(){

	//int pop = 7, col = 5;
	//test_alloc2d();
		
	//////////////////////GEN RANDOM ///////////////////////
	//test_gen_random(pop, col, -100.0, 100.0);
	
	

	///////////////////////OBJFN////////////////////////////
	//hiprandGenerator_t gen;
	//hiprandCreateGenerator(&gen, HIPRAND_RNG_PSEUDO_DEFAULT);
	//hiprandSetPseudoRandomGeneratorSeed(gen,1234ULL);
	//size_t pitch; 
	//test_objectiveFn(pop, col, 4, 3, gen);	
	
	//////////////////////SWAP//////////////////////////////
	//test_swap();
	test_sorted();
	
	return 0;
}
